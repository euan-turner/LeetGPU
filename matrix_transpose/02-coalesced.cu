#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>

template <size_t TILE_DIM>
__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    __shared__ float tile[TILE_DIM][TILE_DIM]; // one element per thread in block

    size_t gx = blockIdx.x * blockDim.x + threadIdx.x; // input column index
    size_t gy = blockIdx.y * blockDim.y + threadIdx.y; // input row index
    size_t lx = threadIdx.x;
    size_t ly = threadIdx.y;


    tile[ly][lx] = (gx < cols && gy < rows) ? input[gy * cols + gx] : 0.0f;
    __syncthreads();

    // 2. warp picks up strided values from tile, but writes them contiguously
    size_t tx = blockIdx.y * TILE_DIM + lx; // output column index
    size_t ty = blockIdx.x * TILE_DIM + ly; // output row index
    if (tx < rows && ty < cols) {
        output[ty * rows + tx] = tile[lx][ly];
    }
}

extern "C" void solve(const float* input, float* output, int rows, int cols) {
    const size_t BLOCK_SIZE = 16;
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE,
                       (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrix_transpose_kernel<BLOCK_SIZE><<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}

// Host-side correctness checker
bool check_transpose(const float* input, const float* output, int rows, int cols) {
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c) {
            float expected = input[r * cols + c];
            float got = output[c * rows + r];
            if (expected != got) {
                printf("Mismatch at input[%d][%d], output[%d][%d]: expected %f, got %f\n",
                       r, c, c, r, expected, got);
                return false;
            }
        }
    }
    return true;
}

int main() {
    int rows = 256;
    int cols = 128;
    size_t size = rows * cols * sizeof(float);

    // Host memory
    float* h_input  = (float*)malloc(size);
    float* h_output = (float*)malloc(rows * cols * sizeof(float));

    // Initialize input matrix with some values
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c) {
            h_input[r * cols + c] = static_cast<float>(r * cols + c + 1);
        }
    }

    // Device memory
    float* d_input;
    float* d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, rows * cols * sizeof(float));

    // Copy input to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Launch transpose
    solve(d_input, d_output, rows, cols);

    // Copy result back
    hipMemcpy(h_output, d_output, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // Check correctness
    bool correct = check_transpose(h_input, h_output, rows, cols);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    // if (!correct) return -1;
    return 0;
}
